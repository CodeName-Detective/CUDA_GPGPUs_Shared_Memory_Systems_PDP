#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <functional>
#include <random>
#include <chrono>

#define TILE_WIDTH 1024


void error_check(hipError_t& err, int line_no, const char* file_name){
    if (err!=hipSuccess) {
        std::cout<<hipGetErrorString(err)<<" at line "<<line_no<<" in file "<<file_name<<std::endl;
    }
}


__global__ void mat_vec_mul_kernel(float* M, float* V, float* Out, unsigned int num_rows, unsigned int num_cols){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx<num_rows){
        Out[idx] = 0;
        for(int i=0; i<num_cols; ++i){
            Out[idx] += M[idx*num_cols+i]*V[i];
        }
    }
}


__global__ void mat_vec_mul_kernel_tiled(float* M, float* V, float* Out, unsigned int num_rows, unsigned int num_cols){

    // Creating Buffers in shared memory to store data.
    __shared__ float Vs[TILE_WIDTH];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    float inner_product=0.0f;
    for(int phase=0; phase<(num_cols+TILE_WIDTH-1)/TILE_WIDTH; ++phase){
        if((phase*TILE_WIDTH+threadIdx.x)<num_cols){
            Vs[threadIdx.x] = V[phase*TILE_WIDTH+threadIdx.x];
        }
        else{
            Vs[threadIdx.x] = 0.0f;
        }
        __syncthreads(); //Synchronization

        for(int i=0; i<TILE_WIDTH; ++i){
            // To Prevent Out-of-bound memory access for M
            if(phase*TILE_WIDTH+i<num_cols){
                inner_product += M[idx*num_cols+phase*TILE_WIDTH+i]*Vs[i];
            }
        }
        __syncthreads(); //Synchronization
    }

    if(idx<num_rows){
        Out[idx] = inner_product;
    }
}

void mat_vec_mul(float* M_h, float* V_h, float* Out_h, unsigned int& num_rows, unsigned int& num_cols){
    float *M_d, *V_d, *Out_d;

    unsigned int m_size = num_rows*num_cols*sizeof(float), v_size=num_cols*sizeof(float), out_size=num_rows*sizeof(float);

    hipError_t err ;

    // Allocating CUDA Memory
    err = hipMalloc((void **)&M_d, m_size);
    error_check(err, __LINE__, __FILE__);
    err = hipMalloc((void **)&V_d, v_size);
    error_check(err, __LINE__, __FILE__);
    err = hipMalloc((void **)&Out_d, out_size);
    error_check(err, __LINE__, __FILE__);

    // Copying Data From Host To Device
    err = hipMemcpy(M_d, M_h, m_size, hipMemcpyHostToDevice);
    error_check(err, __LINE__, __FILE__);
    err = hipMemcpy(V_d, V_h, v_size, hipMemcpyHostToDevice);
    error_check(err, __LINE__, __FILE__);

    // Calling Kernel
    /*mat_vec_mul_kernel<<<(int)std::ceil(num_rows/256.0f), 256>>>(M_d, V_d, Out_d, num_rows, num_cols);*/
    mat_vec_mul_kernel_tiled<<<(num_rows+TILE_WIDTH-1)/TILE_WIDTH), TILE_WIDTH>>>(M_d, V_d, Out_d, num_rows, num_cols);
    err = hipGetLastError();
    error_check(err, __LINE__, __FILE__);

    // Copying Output from Device to Host
    err = hipMemcpy(Out_h, Out_d, out_size, hipMemcpyDeviceToHost);
    error_check(err, __LINE__, __FILE__);

    // Freeing CUDA Memeory
    err = hipFree(M_d);
    error_check(err, __LINE__, __FILE__);
    err = hipFree(V_d);
    error_check(err, __LINE__, __FILE__);
    err = hipFree(Out_d);
    error_check(err, __LINE__, __FILE__);
}


int main(){
    unsigned int num_rows, num_cols;

    std::cout<<"Enter the Number of rows:";
    std::cin>>num_rows;

    std::cout<<"Enter the Number of columns:";
    std::cin>>num_cols;

    std::vector<float> M(num_rows*num_cols), V(num_cols), Out(num_rows);

    // Generate Random Numbers
    std::mt19937 gen(666);
    std::normal_distribution<float> N(-1.0, 1.0);
    std::generate(std::begin(M), std::end(M), std::bind(N, gen));
    std::generate(std::begin(V), std::end(V), std::bind(N, gen));

    auto t0 = std::chrono::system_clock::now();

    mat_vec_mul(M.data(), V.data(), Out.data(), num_rows, num_cols);

    auto t1 = std::chrono::system_clock::now();
    auto elapsed_par = std::chrono::duration<double>(t1 - t0);
    std::cout <<"Elapsed time is:"<<elapsed_par.count() << std::endl;

    return 0;
}